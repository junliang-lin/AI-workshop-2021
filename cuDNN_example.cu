#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <time.h>

#ifdef _WIN32
using uint = unsigned int;
using uchar = unsigned char;
using ushort = unsigned short;
using int64_t = long long;
using uint64_t = unsigned long long;
#else
#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long
#endif


#define CUDA_CALL(f) { \
    hipError_t err = (f); \
    if (err != hipSuccess) { \
      std::cout \
          << "    Error occurred: " << err << std::endl; \
      std::exit(1); \
    } \
  }
  
#define CUDNN_CALL(f) { \
cudnnStatus_t err = (f); \
if (err != CUDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << std::endl; \
    std::exit(1); \
} \
}

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    std::cout << "NBody.GPU" << std::endl << "=========" << std::endl << std::endl;

    std::cout << "CUDA version:   v" << CUDART_VERSION << std::endl;    
    
    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "CUDA Devices: " << std::endl << std::endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::cout << "  Global memory:   " << props.totalGlobalMem / mb << " MB" << std::endl;
        std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << " KB" << std::endl;
        std::cout << "  Constant memory: " << props.totalConstMem / kb << " KB" << std::endl;
        std::cout << "  Block registers: " << props.regsPerBlock << std::endl << std::endl;

        std::cout << "  Warp size:         " << props.warpSize << std::endl;
        std::cout << "  Threads per block: " << props.maxThreadsPerBlock << std::endl;
        std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << std::endl;
        std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << std::endl;
        std::cout << std::endl;
    }
}


void print(const float *data, int n, int c, int h, int w) {
  std::vector<float> buffer(1 << 20);
  CUDA_CALL(hipMemcpy(
        buffer.data(), data,
        n * c * h * w * sizeof(float),
        hipMemcpyDeviceToHost));
  int a = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < c; ++j) {
      std::cout << "n=" << i << ", c=" << j << ":" << std::endl;
      for (int k = 0; k < h; ++k) {
        for (int l = 0; l < w; ++l) {
          std::cout << std::setw(7) << std::setprecision(5) << std::right << buffer[a];
          ++a;
        }
        std::cout << std::endl;
      }
      break;
    }
    break;
  }
  std::cout << std::endl;
}

__global__ void dev_const(float *px, float k) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = k;
}

__global__ void dev_iota(float *px) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid*0.001;
}

int main(){

  DisplayHeader();
  cudnnHandle_t cudnn;
  CUDNN_CALL(cudnnCreate(&cudnn));

  // input
  const int in_n = 32;
  const int in_c = 32;
  const int in_h = 16;
  const int in_w = 16;
  std::cout << "in_n: " << in_n << std::endl;
  std::cout << "in_c: " << in_c << std::endl;
  std::cout << "in_h: " << in_h << std::endl;
  std::cout << "in_w: " << in_w << std::endl;
  std::cout << std::endl;

  cudnnTensorDescriptor_t in_desc;
  CUDNN_CALL(cudnnCreateTensorDescriptor(&in_desc));
  CUDNN_CALL(cudnnSetTensor4dDescriptor(
        in_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        in_n, in_c, in_h, in_w));

  float *in_data;
  CUDA_CALL(hipMalloc(
        &in_data, in_n * in_c * in_h * in_w * sizeof(float)));

  // filter
  const int filt_k = 64;
  const int filt_c = 32;
  const int filt_h = 3;
  const int filt_w = 3;
  std::cout << "filt_k: " << filt_k << std::endl;
  std::cout << "filt_c: " << filt_c << std::endl;
  std::cout << "filt_h: " << filt_h << std::endl;
  std::cout << "filt_w: " << filt_w << std::endl;
  std::cout << std::endl;

  cudnnFilterDescriptor_t filt_desc;
  CUDNN_CALL(cudnnCreateFilterDescriptor(&filt_desc));
  CUDNN_CALL(cudnnSetFilter4dDescriptor(
        filt_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
        filt_k, filt_c, filt_h, filt_w));

  float *filt_data;
  CUDA_CALL(hipMalloc(
      &filt_data, filt_k * filt_c * filt_h * filt_w * sizeof(float)));

  // convolution
  const int pad_h = 1;
  const int pad_w = 1;
  const int str_h = 1;
  const int str_w = 1;
  const int dil_h = 1;
  const int dil_w = 1;
  std::cout << "pad_h: " << pad_h << std::endl;
  std::cout << "pad_w: " << pad_w << std::endl;
  std::cout << "str_h: " << str_h << std::endl;
  std::cout << "str_w: " << str_w << std::endl;
  std::cout << "dil_h: " << dil_h << std::endl;
  std::cout << "dil_w: " << dil_w << std::endl;
  std::cout << std::endl;

  cudnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(cudnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(cudnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT));

  // CUDNN_CONVOLUTION
  // In this mode, a convolution operation will be done when applying the filter to the images.
  
  // CUDNN_CROSS_CORRELATION
  // In this mode, a cross-correlation operation will be done when applying the filter to the images.

  // output
  int out_n;
  int out_c;
  int out_h;
  int out_w;
  
  CUDNN_CALL(cudnnGetConvolution2dForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        &out_n, &out_c, &out_h, &out_w));
  CUDNN_CALL(cudnnSetConvolutionMathType(conv_desc, CUDNN_TENSOR_OP_MATH));
  // CUDNN_DEFAULT_MATH
  // Tensor Core operations are not used on pre-NVIDIA A100 GPU devices. On A100 GPU architecture devices, Tensor Core TF32 operation is permitted.

  // CUDNN_TENSOR_OP_MATH
  // The use of Tensor Core operations is permitted but will not actively perform datatype down conversion on tensors in order to utilize Tensor Cores.

  // CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION
  // The use of Tensor Core operations is permitted and will actively perform datatype down conversion on tensors in order to utilize Tensor Cores.

  // CUDNN_FMA_MATH
  // Restricted to only kernels that use FMA instructions.

  std::cout << "out_n: " << out_n << std::endl;
  std::cout << "out_c: " << out_c << std::endl;
  std::cout << "out_h: " << out_h << std::endl;
  std::cout << "out_w: " << out_w << std::endl;
  std::cout << std::endl;

  cudnnTensorDescriptor_t out_desc;
  CUDNN_CALL(cudnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(cudnnSetTensor4dDescriptor(
        out_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));

  float *out_data_ref;
  CUDA_CALL(hipMalloc(
        &out_data_ref, out_n * out_c * out_h * out_w * sizeof(float)));
  
  float *out_data;
  CUDA_CALL(hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(float)));

  cudnnConvolutionFwdAlgo_t algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  // CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM
  // This algorithm expresses the convolution as a matrix product without actually explicitly forming the matrix that holds the input tensor data.
  
  // CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM
  // This algorithm expresses convolution as a matrix product without actually explicitly forming the matrix that holds the input tensor data, but still needs some memory workspace to precompute some indices in order to facilitate the implicit construction of the matrix that holds the input tensor data.
  
  // CUDNN_CONVOLUTION_FWD_ALGO_GEMM
  // This algorithm expresses the convolution as an explicit matrix product. A significant memory workspace is needed to store the matrix that holds the input tensor data.

  std::cout << "Convolution algorithm: " << algo << std::endl;
  std::cout << std::endl;

  // workspace
  size_t ws_size;
  CUDNN_CALL(cudnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

  float *ws_data;
  CUDA_CALL(hipMalloc(&ws_data, ws_size));

  std::cout << "Workspace size: " << ws_size << std::endl;
  std::cout << std::endl;

  float alpha = 1.f;
  float beta = 0.f;

  dim3 DimGrid(1, 1, 512);
  dim3 DimBlock(16, 8, 1);

  // perform
  dev_const<<<in_n * in_c, in_w * in_h>>>(in_data, 1.f);
  dev_const<<<filt_k * filt_c, filt_w * filt_h>>>(filt_data, 0.f);

  float GPUtime;
  for(int i=0; i<10; i++){
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop); 
    hipEventRecord(start, 0); 
    CUDNN_CALL(cudnnConvolutionForward(
        cudnn,
        &alpha, in_desc, in_data, filt_desc, filt_data,
        conv_desc, algo, ws_data, ws_size,
        &beta, out_desc, out_data_ref));
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);
    hipEventElapsedTime(&GPUtime, start, stop); 
  }
  // print(out_data_ref, out_n, out_c, out_h, out_w);
  printf("Compute time on GPU with CuDNN: %3.6f ms \n", GPUtime);

  // finalizing
  CUDA_CALL(hipFree(ws_data));
  CUDA_CALL(hipFree(out_data));
  CUDNN_CALL(cudnnDestroyTensorDescriptor(out_desc));
  CUDNN_CALL(cudnnDestroyConvolutionDescriptor(conv_desc));
  CUDA_CALL(hipFree(filt_data));
  CUDNN_CALL(cudnnDestroyFilterDescriptor(filt_desc));
  CUDA_CALL(hipFree(in_data));
  CUDNN_CALL(cudnnDestroyTensorDescriptor(in_desc));
  CUDNN_CALL(cudnnDestroy(cudnn));

  return 0;
}

// compile and run
// nvcc -lcudnn -O3 --gpu-architecture=sm_86 -lineinfo cuDNN_example.cu && ./a.out

// profile tensor core usage
// sudo /usr/local/cuda/bin/nv-nsight-cu-cli --csv --log-file file.csv --metrics sm__pipe_tensor_op_hmma_cycles_active.avg.pct_of_peak_sustained_active  ./work/test/a.out
